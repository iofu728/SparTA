
#include <assert.h>
// CUDA runtime
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <assert.h>
#include <fstream>
#include <iostream>
#include <string>
#include <sstream>
#include <vector>
#include <time.h>
// #include <math>
#include <algorithm>
#include <assert.h>
#include "iostream"
#include "sstream"
#include "time.h"
#include "memory"
#include "vector"
using namespace std;

// #include "utils.hpp"
using namespace std;
#define OFFSET(row, col, ld) ((row) * ld + col)
#define FETCH_FLOAT4(pointer) (reinterpret_cast<float4*>(&pointer))[0]
#define FETCH_UINT32(pointer) (reinterpret_cast<unsigned int*>(&(pointer))[0])
#define FETCH_UINT4(pointer) (reinterpret_cast<uint4*>(&(pointer))[0])
#define FETCH_INT4(pointer) (reinterpret_cast<int4*>(&(pointer))[0])
#define FETCH_INT32(pointer) (reinterpret_cast<int*>(&(pointer))[0])
#define MAX_BLOCK_THREAD_COUNT 1024
#define FULL_MASK 0xffffffff

#define CUBLAS_SAFE_CALL(func)                                                                  \
    do                                                                                          \
    {                                                                                           \
        hipblasStatus_t e = (func);                                                              \
        if (e != HIPBLAS_STATUS_SUCCESS)                                                         \
        {                                                                                       \
            std::stringstream safe_call_ss;                                                     \
            safe_call_ss << "\nerror: " #func " failed with error"                              \
                         << "\nfile: " << __FILE__ << "\nline: " << __LINE__ << "\nmsg: " << e; \
            throw std::runtime_error(safe_call_ss.str());                                       \
        }                                                                                       \
    } while (0)

#define CUDA_SAFE_CALL(x)                                                                         \
    do                                                                                            \
    {                                                                                             \
        hipError_t result = (x);                                                                 \
        if (result != hipSuccess)                                                                \
        {                                                                                         \
            const char *msg = hipGetErrorString(result);                                         \
            std::stringstream safe_call_ss;                                                       \
            safe_call_ss << "\nerror: " #x " failed with error"                                   \
                         << "\nfile: " << __FILE__ << "\nline: " << __LINE__ << "\nmsg: " << msg; \
            throw std::runtime_error(safe_call_ss.str());                                         \
        }                                                                                         \
    } while (0)

__device__ void warpReduce(volatile int* sdata, int tid) {
    sdata[tid] += sdata[tid + 32]; 
    sdata[tid] += sdata[tid + 16]; 
    sdata[tid] += sdata[tid + 8]; 
    sdata[tid] += sdata[tid + 4]; 
    sdata[tid] += sdata[tid + 2]; 
    sdata[tid] += sdata[tid + 1]; 
}

__device__ __forceinline__ const int* add_ptr_u(const int* src, int offset)      \
{                                                                            \
    const int* dst;                                                            \
    asm("{                       \n\t"                                       \
        ".reg .u32 lo,hi,of;     \n\t"                                       \
        "mul.lo.u32 of, %2, %3;  \n\t"                                       \
        "mov.b64    {lo,hi}, %1; \n\t"                                       \
        "add.cc.u32  lo,lo,  of; \n\t"                                       \
        "addc.u32    hi,hi,  0;  \n\t"                                       \
        "mov.b64 %0, {lo,hi};    \n\t"                                       \
        "}" : "=l"(dst) : "l"(src), "r"(offset), "r"((int)sizeof(*src)));    \
    return dst;                                                              \
}

__device__ __forceinline__ const float* add_ptr_f(const float* src, int offset)      \
{                                                                            \
    const float* dst;                                                            \
    asm("{                       \n\t"                                       \
        ".reg .u32 lo,hi,of;     \n\t"                                       \
        "mul.lo.u32 of, %2, %3;  \n\t"                                       \
        "mov.b64    {lo,hi}, %1; \n\t"                                       \
        "add.cc.u32  lo,lo,  of; \n\t"                                       \
        "addc.u32    hi,hi,  0;  \n\t"                                       \
        "mov.b64 %0, {lo,hi};    \n\t"                                       \
        "}" : "=l"(dst) : "l"(src), "r"(offset), "r"((int)sizeof(*src)));    \
    return dst;                                                              \
}

__device__ __forceinline__ float2  _add(float2 x, float2 y) { float2 res; res.x = x.x + y.x; res.y = x.y + y.y; return res; }

void init(float * ptr, size_t length, float sparsity)
{
    for (int i = 0; i < length; i++)
    {
        float pro = static_cast<float>(rand()) / static_cast<float>(RAND_MAX);
        if (pro < sparsity)
        {
            ptr[i] = 0.0;
        }
        else
        {
            // ptr[i] = static_cast<float>(rand()) / static_cast<float>(RAND_MAX);
            ptr[i] = 1;
        }
    }
}

void calculate_reference(int m, int k, int n, float * A, float *B, float * C) 
{
    for(int i=0; i<m; i++){
        for(int j=0; j<n; j++){
            float sum = 0.0;
            for(int tmp=0; tmp<k; tmp++){
                sum += A[i * k + tmp] * B[tmp * n + j];
            }
            C[i*n+j] = sum;
        }
    }
}
template <
    const int N_TILE_SIZE,
    const int BLOCK_SIZE_K,
    const int BLOCK_SIZE_N
>
__global__ void FINEGRAINED_CONDENSE_KERNEL(const int* __restrict__  csr_row, const int* __restrict__  csr_col, const float* __restrict__  csr_val,  float* __restrict__  B, float* __restrict__  C, const int M, const int K, const int N){
    

    int by = blockIdx.y;
    int bx = blockIdx.x;
    int tid = threadIdx.x;
    // int ty = threadIdx.y;
    // int tx = threadIdx.x;
    const int padding = 1;
    __shared__ int Is[BLOCK_SIZE_K];
    __shared__ float Vs[BLOCK_SIZE_K];
    __shared__ float Bs[BLOCK_SIZE_K*(BLOCK_SIZE_N+padding)];
    assert(N_TILE_SIZE%BLOCK_SIZE_N==0);
    int index_start = csr_row[by];
    int index_end = csr_row[by+1];
    int row_nnz = index_end - index_start;
    int n_thread_per_row = BLOCK_SIZE_N/4;
    int n_stride = blockDim.x/n_thread_per_row;
    int ty = tid/n_thread_per_row;
    int tx = tid%n_thread_per_row;
    #pragma unroll
    for(int n_round=0; n_round<N_TILE_SIZE/BLOCK_SIZE_N; n_round++){
        float sum = 0;
        int n_start = bx * N_TILE_SIZE + n_round * BLOCK_SIZE_N;
        int n_end = n_start + BLOCK_SIZE_N;
        #pragma unroll
        for(int k_round=0; k_round< (index_end-index_start-1+BLOCK_SIZE_K)/BLOCK_SIZE_K; k_round++){
            // load the A to the shared memory
            int k_start = index_start + k_round * BLOCK_SIZE_K;
            // int k_end = min(k_start+ BLOCK_SIZE_K, index_end);
            int k_end = k_start+ BLOCK_SIZE_K;
            for(int _pos=tid+k_start; _pos<k_end; _pos+=blockDim.x){
                if(_pos<index_end){
                    Is[_pos-k_start] = csr_col[_pos];
                    Vs[_pos-k_start] = csr_val[_pos];
                }else{
                    Vs[_pos-k_start] = 0;
                }
            }
            __syncthreads();
            // load B to the shared memory
            #pragma unroll
            for(int _pos=ty; _pos<min(index_end-k_start, BLOCK_SIZE_K); _pos+=n_stride){
                int k_offset = Is[_pos];
                FETCH_FLOAT4(Bs[OFFSET(_pos, tx*4, BLOCK_SIZE_N)]) = 
                    FETCH_FLOAT4(B[OFFSET(k_offset, n_start+tx*4, N)]);
            }
            __syncthreads();
            // computation the spmv
            #pragma unroll
            for(int i=0;i<BLOCK_SIZE_K;i++){
                sum += Vs[i]*Bs[OFFSET(i, tid,BLOCK_SIZE_N)];
            }

        }
        // write backto C
        C[OFFSET(by, n_start+tid, N)] = sum;
    }

}

void FINEGRAINED_CONDESE(int *csr_row, int * csr_col, float* csr_val, float * B, float* C, int M, int K, int N)
{
    const int N_TILE_SIZE = 1024;
    const int BLOCK_SIZE_N = 256;
    const int BLOCK_SIZE_K = 4;
    dim3 gridDim(N/N_TILE_SIZE, M);
    dim3 blockDim(BLOCK_SIZE_N);
    FINEGRAINED_CONDENSE_KERNEL<N_TILE_SIZE, BLOCK_SIZE_K, BLOCK_SIZE_N><<<gridDim, blockDim>>>(csr_row, csr_col, csr_val, B, C, M, K, N);

}


template <
    const int BLOCK_SIZE_M,
    const int BLOCK_SIZE_K,
    const int BLOCK_SIZE_N
>
__global__ void FINEGRAINED_CONDENSE_KERNEL_V2(const int* __restrict__  csr_row, const int* __restrict__  csr_col, const float* __restrict__  csr_val,  float* __restrict__  B, float* __restrict__  C, const int M, const int K, const int N){
    

    int by = blockIdx.y;
    int bx = blockIdx.x;
    int tid = threadIdx.x;
    // int ty = threadIdx.y;
    // int tx = threadIdx.x;
    const int padding = 1;

    __shared__ int Is[BLOCK_SIZE_M * BLOCK_SIZE_K];
    __shared__ float Vs[BLOCK_SIZE_M * BLOCK_SIZE_K];
    int ty = tid/BLOCK_SIZE_N;
    int tx = tid%BLOCK_SIZE_N;
    int row_id = by * BLOCK_SIZE_M + ty;
    int index_start = csr_row[row_id];
    int index_end = csr_row[row_id+1];
    const int n_thread_per_row = BLOCK_SIZE_N;

    #pragma unroll
    // for(int n_round=0; n_round<N_TILE_SIZE/BLOCK_SIZE_N; n_round++){
    float sum = 0;
    int n_start = bx * BLOCK_SIZE_N;
    // int n_end = n_start + BLOCK_SIZE_N;
    #pragma unroll
    for(int k_round=0; k_round< (index_end-index_start-1+BLOCK_SIZE_K)/BLOCK_SIZE_K; k_round++){
        // load the A to the shared memory
        int k_start = index_start + k_round * BLOCK_SIZE_K;
        // int k_end = min(k_start+ BLOCK_SIZE_K, index_end);
        int k_end = k_start + BLOCK_SIZE_K;
        for(int _pos=tx+k_start; _pos<k_end; _pos+=n_thread_per_row){
            if(_pos<index_end){
                Is[ty*BLOCK_SIZE_K + _pos-k_start] = csr_col[_pos];
                Vs[ty*BLOCK_SIZE_K + _pos-k_start] = csr_val[_pos];
            }else{
                Vs[ty*BLOCK_SIZE_K + _pos-k_start] = 0;
            }
        }
        __syncthreads();
        // load B to the shared memory
        // #pragma unroll
        // for(int _pos=ty; _pos<min(index_end-k_start, BLOCK_SIZE_K); _pos+=n_stride){
        //     int k_offset = Is[_pos];
        //     FETCH_FLOAT4(Bs[OFFSET(_pos, tx*4, BLOCK_SIZE_N)]) = 
        //         FETCH_FLOAT4(B[OFFSET(k_offset, n_start+tx*4, N)]);
        // }
        // __syncthreads();
        // computation the spmv
        #pragma unroll
        for(int i=0;i<BLOCK_SIZE_K;i++){
            int k_offset = Is[ty*BLOCK_SIZE_K+i];
            sum += Vs[ty*BLOCK_SIZE_K + i]*B[OFFSET(k_offset, n_start + tx, N)];
        }

    }
    // write backto C
    C[OFFSET(row_id, n_start+tx, N)] = sum;
    // }

}
void FINEGRAINED_CONDESE_V2(int *csr_row, int * csr_col, float* csr_val, float * B, float* C, int M, int K, int N)
{
    const int BLOCK_SIZE_M = 32;
    const int BLOCK_SIZE_N = 32;
    const int BLOCK_SIZE_K = 32;
    dim3 gridDim(N/BLOCK_SIZE_N, M/BLOCK_SIZE_M);
    dim3 blockDim(BLOCK_SIZE_M*BLOCK_SIZE_N);
    FINEGRAINED_CONDENSE_KERNEL_V2<BLOCK_SIZE_M, BLOCK_SIZE_K, BLOCK_SIZE_N><<<gridDim, blockDim>>>(csr_row, csr_col, csr_val, B, C, M, K, N);

}

template <
    const int BLOCK_SIZE_M,
    const int BLOCK_SIZE_K,
    const int BLOCK_SIZE_N
>
__global__ void FINEGRAINED_CONDENSE_KERNEL_V3(const int* __restrict__  csr_row, const int* __restrict__  csr_col, const float* __restrict__  csr_val,  float* __restrict__  B, float* __restrict__  C, const int M, const int K, const int N){
    

    int by = blockIdx.y;
    int bx = blockIdx.x;
    int tid = threadIdx.x;
    const int padding = 1;

    __shared__ int Is[BLOCK_SIZE_M * BLOCK_SIZE_K];
    __shared__ float Vs[BLOCK_SIZE_M * BLOCK_SIZE_K];

    int tx = tid % BLOCK_SIZE_N;
    const int n_thread_per_row = BLOCK_SIZE_N;
    // for(int n_round=0; n_round<N_TILE_SIZE/BLOCK_SIZE_N; n_round++){
    float sum = 0;
    int n_start = bx * BLOCK_SIZE_N;
    int m_stride = blockDim.x / BLOCK_SIZE_N;
    #pragma unroll
    for(int ty=tid/BLOCK_SIZE_N; ty<BLOCK_SIZE_M; ty+=m_stride){
        sum = 0;
        int row_id = by * BLOCK_SIZE_M + ty;
        int index_start = csr_row[row_id];
        int index_end = csr_row[row_id+1];

        #pragma unroll
        for(int k_round=0; k_round< (index_end-index_start-1+BLOCK_SIZE_K)/BLOCK_SIZE_K; k_round++){
            // load the A to the shared memory
            int k_start = index_start + k_round * BLOCK_SIZE_K;
            // int k_end = min(k_start+ BLOCK_SIZE_K, index_end);
            int k_end = k_start + BLOCK_SIZE_K;
            for(int _pos=tx+k_start; _pos<k_end; _pos+=n_thread_per_row){
                if(_pos<index_end){
                    Is[ty*BLOCK_SIZE_K + _pos-k_start] = csr_col[_pos];
                    Vs[ty*BLOCK_SIZE_K + _pos-k_start] = csr_val[_pos];
                }else{
                    Vs[ty*BLOCK_SIZE_K + _pos-k_start] = 0;
                }
            }
            __syncthreads();

            #pragma unroll
            for(int i=0;i<BLOCK_SIZE_K;i++){
                int k_offset = Is[ty*BLOCK_SIZE_K+i];
                sum += Vs[ty*BLOCK_SIZE_K + i]*B[OFFSET(k_offset, n_start + tx, N)];
            }

        }
        // write backto C
        C[OFFSET(row_id, n_start+tx, N)] = sum;
    }
    // }

}
void FINEGRAINED_CONDESE_V3(int *csr_row, int * csr_col, float* csr_val, float * B, float* C, int M, int K, int N)
{
    const int BLOCK_SIZE_M = 32;
    const int BLOCK_SIZE_N = 64;
    const int BLOCK_SIZE_K = 32;
    dim3 gridDim(N/BLOCK_SIZE_N, M/BLOCK_SIZE_M);
    dim3 blockDim(512);
    FINEGRAINED_CONDENSE_KERNEL_V3<BLOCK_SIZE_M, BLOCK_SIZE_K, BLOCK_SIZE_N><<<gridDim, blockDim>>>(csr_row, csr_col, csr_val, B, C, M, K, N);

}



template <
    const int BLOCK_SIZE_M,
    const int BLOCK_SIZE_K,
    const int BLOCK_SIZE_N
>
__global__ void FINEGRAINED_CONDENSE_KERNEL_V4(const int* __restrict__  csr_row, const int* __restrict__  csr_col, const float* __restrict__  csr_val,  float* __restrict__  B, float* __restrict__  C, const int M, const int K, const int N){
    

    int by = blockIdx.y;
    int bx = blockIdx.x;
    int tid = threadIdx.x;
    const int padding = 1;

    __shared__ int Is[BLOCK_SIZE_M * BLOCK_SIZE_K];
    __shared__ float Vs[BLOCK_SIZE_M * BLOCK_SIZE_K];
    const int n_threads_per_row = 32;
    int ty = tid/n_threads_per_row;
    int tx = tid%n_threads_per_row;
    int row_id = by * BLOCK_SIZE_M + ty;
    int index_start = csr_row[row_id];
    int index_end = csr_row[row_id+1];
    const int n_thread_per_row = BLOCK_SIZE_N;

    #pragma unroll
    // for(int n_round=0; n_round<N_TILE_SIZE/BLOCK_SIZE_N; n_round++){
    float sum = 0;
    int n_start = bx * BLOCK_SIZE_N;
    // int n_end = n_start + BLOCK_SIZE_N;
    #pragma unroll
    for(int k_round=0; k_round< (index_end-index_start-1+BLOCK_SIZE_K)/BLOCK_SIZE_K; k_round++){
        // load the A to the shared memory
        int k_start = index_start + k_round * BLOCK_SIZE_K;
        // int k_end = min(k_start+ BLOCK_SIZE_K, index_end);
        int k_end = k_start + BLOCK_SIZE_K;
        for(int _pos=tx+k_start; _pos<k_end; _pos+=n_thread_per_row){
            if(_pos<index_end){
                Is[ty*BLOCK_SIZE_K + _pos-k_start] = csr_col[_pos];
                Vs[ty*BLOCK_SIZE_K + _pos-k_start] = csr_val[_pos];
            }else{
                Vs[ty*BLOCK_SIZE_K + _pos-k_start] = 0;
            }
        }
        __syncthreads();
        // load B to the shared memory
        // #pragma unroll
        // for(int _pos=ty; _pos<min(index_end-k_start, BLOCK_SIZE_K); _pos+=n_stride){
        //     int k_offset = Is[_pos];
        //     FETCH_FLOAT4(Bs[OFFSET(_pos, tx*4, BLOCK_SIZE_N)]) = 
        //         FETCH_FLOAT4(B[OFFSET(k_offset, n_start+tx*4, N)]);
        // }
        // __syncthreads();
        // computation the spmv
        #pragma unroll
        for(int i=0;i<BLOCK_SIZE_K;i++){
            int k_offset = Is[ty*BLOCK_SIZE_K+i];
            sum += Vs[ty*BLOCK_SIZE_K + i]*B[OFFSET(k_offset, n_start + tx, N)];
        }

    }
    // write backto C
    C[OFFSET(row_id, n_start+tx, N)] = sum;
    // }

}
void FINEGRAINED_CONDESE_V4(int *csr_row, int * csr_col, float* csr_val, float * B, float* C, int M, int K, int N)
{
    const int BLOCK_SIZE_M = 32;
    const int BLOCK_SIZE_N = 32;
    const int BLOCK_SIZE_K = 32;
    dim3 gridDim(N/BLOCK_SIZE_N, M/BLOCK_SIZE_M);
    dim3 blockDim(BLOCK_SIZE_M*BLOCK_SIZE_N);
    FINEGRAINED_CONDENSE_KERNEL_V2<BLOCK_SIZE_M, BLOCK_SIZE_K, BLOCK_SIZE_N><<<gridDim, blockDim>>>(csr_row, csr_col, csr_val, B, C, M, K, N);

}


int convert_csr(float * ptr, int32_t row, int32_t col, int32_t * row_idx, int32_t * col_idx, float * values)
{
    auto v_row_idx = std::make_shared<vector<int32_t>>();
    auto v_col_idx = std::make_shared<vector<int32_t>>();
    auto v_values = std::make_shared<vector<float>>();

    for (int i = 0; i < row; i++)
    {
        v_row_idx->push_back(v_values->size());
        for (int j = 0; j < col; j++)
        {
            size_t pos = i * col + j;
            if (ptr[pos] < 1e-8)
            {
                // sparsity
                continue;
            }
            else
            {
                v_values->push_back(ptr[pos]);
                v_col_idx->push_back(j);
            }
        }
    }
    v_row_idx->push_back(v_values->size());
    int row_idx_size = sizeof(int32_t)*v_row_idx->size();
    int col_idx_size = sizeof(int32_t)*v_col_idx->size();
    int values_size = sizeof(float)*v_values->size();
    printf("values_size: %d\n", values_size);

    memcpy(row_idx, v_row_idx->data(), row_idx_size);
    memcpy(col_idx, v_col_idx->data(), col_idx_size);
    memcpy(values, v_values->data(), values_size);
    return v_values->size();
}


int main()
{
    int M, K, N;
    M = 4096;
    K = 4096;
    N = 4096;
    const int n_iter = 100;
    float sparsity_ratio = 0.6;

    hipEvent_t time_start, time_end;
    CUDA_SAFE_CALL(hipEventCreate(&time_start));
    CUDA_SAFE_CALL(hipEventCreate(&time_end));
    float msecTotal = 0;
    float * A, *B, *C, *val, *refC;
    float * dA, *dB, *dC, *d_val;

    int * mask, *d_mask, *row, *d_row, *row_pos, *d_row_pos, *col, *d_col, *d_extra_buffer;
    A = (float*) malloc(sizeof(float) * M * K);
    B = (float*) malloc(sizeof(float) * K * N);
    C = (float*) malloc(sizeof(float) * M * N);
    refC = (float*) malloc(sizeof(float) * M * N);

    row = (int*) malloc(sizeof(int) * (M+1));
    col = (int*) malloc(sizeof(int) *  M * K);
    val = (float*) malloc(sizeof(float) * M * K);
    init(A, M*K, sparsity_ratio);
    init(B, N*K, 0);
    // apply mask

    convert_csr(A, M, K, row, col, val);
    int nnz = row[M];
    
    printf("NNZ: %d\n", nnz);
    printf("Sparsity ratio: %f\n", 1-nnz*1.0/M/K);
    CUDA_SAFE_CALL(hipMalloc(&d_mask, sizeof(int) * M * K));
    CUDA_SAFE_CALL(hipMalloc(&d_row, sizeof(int) * (M + 1)));
    CUDA_SAFE_CALL(hipMalloc(&d_col, sizeof(int) * M * K));

    CUDA_SAFE_CALL(hipMalloc(&d_val, sizeof(float) * M * K));
    CUDA_SAFE_CALL(hipMalloc(&dA, sizeof(float) * M * K));
    CUDA_SAFE_CALL(hipMalloc(&dB, sizeof(float) * N * K));
    CUDA_SAFE_CALL(hipMalloc(&dC, sizeof(float) * M * N));
    CUDA_SAFE_CALL(hipMemset(dC, 0, sizeof(float)* M * N));
    
    CUDA_SAFE_CALL(hipMemcpy(dA, A, sizeof(float)*M*K, hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(dB, B, sizeof(float)*K*N, hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_row, row, sizeof(int)*(M+1), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_col, col, sizeof(int)* M * K, hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_val, val, sizeof(float) * M * K, hipMemcpyHostToDevice));

    

    // KxM = KxN * (MxN)^T
    CUDA_SAFE_CALL(hipEventRecord(time_start));

    for(int run=0; run<n_iter; run++){
        // FINEGRAINED_CONDESE_V2(d_row, d_col, d_val, dB, dC, M, K, N);
        FINEGRAINED_CONDESE_V3(d_row, d_col, d_val, dB, dC, M, K, N);
    }
    CUDA_SAFE_CALL(hipEventRecord(time_end));
    CUDA_SAFE_CALL(hipEventSynchronize(time_end));
    CUDA_SAFE_CALL(hipEventElapsedTime(&msecTotal, time_start, time_end));
    printf("Time Cost: %.3fms\n", msecTotal/n_iter);
    CUDA_SAFE_CALL(hipMemcpy(C, dC, sizeof(float) * M * N, hipMemcpyDeviceToHost));
    calculate_reference(M, K, N, A, B, refC);
    for(int i=0;i<M*N;i++){
        if(fabs(C[i]-refC[i])/fabs(refC[i])>0.001)
            printf("%f %f\n", C[i], refC[i]);
    }


    return 0;

}
