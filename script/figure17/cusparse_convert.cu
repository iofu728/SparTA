#include "hipsparse.h"
#include "iostream"
#include "sstream"
#include "hip/hip_runtime.h"
#include "time.h"
#include "memory"
#include "hipblas.h"
#include "vector"
#include "utils.hpp"
using namespace std;

using namespace std;
// Macro definition for the cuda and cusparse
// cuSparse SPMM interface

#define CUBLAS_SAFE_CALL(func)                                                                  \
    do                                                                                          \
    {                                                                                           \
        hipblasStatus_t e = (func);                                                              \
        if (e != HIPBLAS_STATUS_SUCCESS)                                                         \
        {                                                                                       \
            std::stringstream safe_call_ss;                                                     \
            safe_call_ss << "\nerror: " #func " failed with error"                              \
                         << "\nfile: " << __FILE__ << "\nline: " << __LINE__ << "\nmsg: " << e; \
            throw std::runtime_error(safe_call_ss.str());                                       \
        }                                                                                       \
    } while (0)
#define CUSPARSE_SAFE_CALL(func)                                                                \
    do                                                                                          \
    {                                                                                           \
        hipsparseStatus_t e = (func);                                                            \
        if (e != HIPSPARSE_STATUS_SUCCESS)                                                       \
        {                                                                                       \
            std::stringstream safe_call_ss;                                                     \
            safe_call_ss << "\nerror: " #func " failed with error"                              \
                         << "\nfile: " << __FILE__ << "\nline: " << __LINE__ << "\nmsg: " << e; \
            throw std::runtime_error(safe_call_ss.str());                                       \
        }                                                                                       \
    } while (0)

#define CUDA_SAFE_CALL(x)                                                                         \
    do                                                                                            \
    {                                                                                             \
        hipError_t result = (x);                                                                 \
        if (result != hipSuccess)                                                                \
        {                                                                                         \
            const char *msg = hipGetErrorString(result);                                         \
            std::stringstream safe_call_ss;                                                       \
            safe_call_ss << "\nerror: " #x " failed with error"                                   \
                         << "\nfile: " << __FILE__ << "\nline: " << __LINE__ << "\nmsg: " << msg; \
            throw std::runtime_error(safe_call_ss.str());                                         \
        }                                                                                         \
    } while (0)

int cusparse_csr_convert(
    float* dense_value,
    int n_row,
    int n_col,
    int * csr_row,
    int * csr_col,
    float * csr_val)
{
    hipsparseHandle_t     handle = NULL;
    hipsparseSpMatDescr_t matB;
    hipsparseDnMatDescr_t matA;
    static void*                dBuffer    = NULL;
    size_t               bufferSize = 0;
    CUSPARSE_SAFE_CALL(hipsparseCreate(&handle));

    CUSPARSE_SAFE_CALL(hipsparseCreateDnMat(&matA, n_row, n_col, n_col, dense_value,
                                    HIP_R_32F, HIPSPARSE_ORDER_ROW));
    CUSPARSE_SAFE_CALL( hipsparseCreateCsr(&matB, n_row, n_col, 0,
                                    csr_row, NULL, NULL,
                                    HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                    HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) );
    CUSPARSE_SAFE_CALL( hipsparseDenseToSparse_bufferSize(
                                        handle, matA, matB,
                                        HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT,
                                        &bufferSize) );
    if (dBuffer == NULL)
        CUDA_SAFE_CALL( hipMalloc(&dBuffer, bufferSize) );
    CUSPARSE_SAFE_CALL( hipsparseDenseToSparse_analysis(handle, matA, matB,
                                        HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT,
                                        dBuffer) );
    int64_t num_rows_tmp, num_cols_tmp, nnz;
    CUSPARSE_SAFE_CALL( hipsparseSpMatGetSize(matB, &num_rows_tmp, &num_cols_tmp,
                                        &nnz) );
    // torch::Tensor csr_col = torch::empty_like({nnz}, csr_row);
    // torch::Tensor csr_values = torch::empty_like({nnz}, dense_values);
    CUSPARSE_SAFE_CALL( hipsparseCsrSetPointers(matB, csr_row, csr_col, csr_val) );
    // execute Sparse to Dense conversion
    CUSPARSE_SAFE_CALL( hipsparseDenseToSparse_convert(handle, matA, matB,
                                        HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT,
                                        dBuffer) );
    CUSPARSE_SAFE_CALL( hipsparseDestroyDnMat(matA) );
    CUSPARSE_SAFE_CALL( hipsparseDestroySpMat(matB) );
    CUSPARSE_SAFE_CALL( hipsparseDestroy(handle) );
    return 0;
}


int main(int argc, char *argv[]){
    float sparsity_ratio = atof(argv[1]);
    printf("Sparsity Ratio=%f\n", sparsity_ratio);
    // Calculate the matA(Activation: Shape=mxk) * matB(Weight:Shape=k*n)
    // Specify the random seed here
    srand(1);
    int32_t * row_idx, *col_idx, *d_row, *d_col;
    int nnz;
    float * values, *d_val;
    float * matA, *matB, *matC, *matC_ref,*d_matA, *d_matB, *d_matC, *dBuffer;
    const int m = atoi(argv[2]);
    const int k = atoi(argv[3]);
    
    //int m=1024, k=1024, n=1024;
    float alpha=1.0, beta=0.0;
    float sparsity = sparsity_ratio;

    matA = (float*) malloc(sizeof(float)*m*k);
    
    init(matA, m*k, sparsity_ratio);

    CUDA_SAFE_CALL(hipMalloc(&d_matA, sizeof(float)*m*k));
    CUDA_SAFE_CALL(hipMalloc(&d_row, sizeof(float)*m*k));
    CUDA_SAFE_CALL(hipMalloc(&d_col, sizeof(float)*m*k));
    CUDA_SAFE_CALL(hipMalloc(&d_val, sizeof(float)*m*k));
    CUDA_SAFE_CALL(hipMemcpy(d_matA, matA, sizeof(float)*m*k, hipMemcpyHostToDevice));
   
    hipEvent_t start, stop;
    CUDA_SAFE_CALL(hipEventCreate(&start));
    CUDA_SAFE_CALL(hipEventCreate(&stop));
    float msecTotal = 0;
    int nIter = 3000;
    


    CUDA_SAFE_CALL(hipEventRecord(start));

    for(int i = 0; i < nIter; i += 1){
        cusparse_csr_convert(d_matA, m, k, d_row, d_col, d_val);
    }

    CUDA_SAFE_CALL(hipEventRecord(stop));
    CUDA_SAFE_CALL(hipEventSynchronize(stop));
    CUDA_SAFE_CALL(hipEventElapsedTime(&msecTotal, start, stop));

    float msecPerMatrixMul = msecTotal / nIter;
    printf("Time= %f msec\n", msecPerMatrixMul);

    return 0;
}